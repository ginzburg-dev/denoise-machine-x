#include "hip/hip_runtime.h"
#include <dmxdenoiser/DMXImage.hpp>
#include <dmxdenoiser/DMXImageView.hpp>
#include <dmxdenoiser/Kernel2D.hpp>
#include <dmxdenoiser/Pixel.hpp>
#include <dmxdenoiser/utils/NumericUtils.hpp>

#include <hip/hip_runtime.h>

namespace dmxdenoiser
{

    __global__ void convolve2D_CUDA_kernel(DMXImageView in, DMXImageView out, int* frames, int framesSize, int* layers, int layersSize,
                                    float* kernel, int kernelSize, float strength, bool filterAlpha) 
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;
        int s = blockIdx.z;

        int frameIdx = layers[s / layersSize];
        int layerIdx = layers[s % layersSize];
        int frame = frames[frameIdx];
        int layer = layers[layerIdx];
        int offset = kernelSize/2;

        PixelRGBA orig = in.get(x, y, frame, layer);
        PixelRGBA sum = {0.0f, 0.0f, 0.0f, 0.0f};
        for(int ky = -offset; ky <= offset; ++ky)
            for(int kx = -offset; kx <= offset; ++kx)
            {
                int px = clampf(x + kx, 0, in.width - 1);
                int py = clampf(y + ky, 0, in.height - 1);
                sum += kernel[(ky + offset)*kernelSize + (kx + offset)] * in.get(px, py, frame, layer);
            }
        sum = blendPixels(orig, sum, strength, filterAlpha);
        out.at(x, y, frame, layer) = sum;
    }

    void convolve2D_CUDA(const DMXImage& in_, DMXImage& out_, std::vector<int> frames_, std::vector<int> layers_,
                                    const Kernel2D& kernel_, float strength, bool filterAlpha)
    {
        int framesSize = frames_.size();
        int* frames = new int[framesSize];
        int layersSize = layers_.size();
        int* layers = new int[layersSize];
        

        convolve2D_CUDA_kernel<<<>>>
    }

} // namespace dmxdenoiser
