#include "hip/hip_runtime.h"
#include <dmxdenoiser/filters/ConvolutionCUDA.hpp>
#include <dmxdenoiser/DMXImage.hpp>
#include <dmxdenoiser/DMXImageView.hpp>
#include <dmxdenoiser/Kernel2D.hpp>
#include <dmxdenoiser/utils/NumericUtils.hpp>
#include <dmxdenoiser/Pixel.hpp>

#include <cstddef>

#include <hip/hip_runtime.h>

#define CUDA_CHECK(x) do { hipError_t _e = (x); if (_e != hipSuccess) \
  throw std::runtime_error(std::string("CUDA error: ")+hipGetErrorString(_e)); } while(0)

namespace dmxdenoiser
{

    __global__ void convolve2D_CUDA_kernel(DMXImageView in, DMXImageView out, int* frames, int framesSize, int* layers, int layersSize,
                                    float* kernel, int kernelSize, float strength, bool filterAlpha) 
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;
        int s = blockIdx.z;

        if (x >= in.width || y >= in.height) return;

        const int total = framesSize * layersSize;
        if (s >= total) return;

        int frameIdx = s / layersSize;
        int layerIdx = s % layersSize;
        int frame = frames[frameIdx];
        int layer = layers[layerIdx];
        int offset = kernelSize >> 1;

        float* orig = in.at(x, y, frame, layer);
        float sum_r = 0.0f; float sum_g = 0.0f; float sum_b = 0.0f; float sum_a = 0.0f; 
        for(int ky = -offset; ky <= offset; ++ky) 
        {
            int py = clampi(y + ky, 0, in.height - 1);
            int krow = (ky + offset)*kernelSize;
            for(int kx = -offset; kx <= offset; ++kx)
            {
                int px = clampi(x + kx, 0, in.width - 1);
                float w = kernel[krow + (kx + offset)];
                float* p = in.at(px, py, frame, layer);
                sum_r += w * p[0];
                sum_g += w * p[1];
                sum_b += w * p[2];
                sum_a += w * p[3];
            }
        }
        float out_r = floatsBlend(orig[0], sum_r, strength);
        float out_g = floatsBlend(orig[1], sum_g, strength);
        float out_b = floatsBlend(orig[2], sum_b, strength);
        float out_a = filterAlpha ? floatsBlend(orig[3], sum_a, strength) : orig[3];

        float* dist = out.at(x, y, frame, layer);
        dist[0] = out_r; dist[1] = out_g; dist[2] = out_b; dist[3] = out_a;
    }

    void convolve2D_CUDA(const DMXImage& in_, DMXImage& out_, const std::vector<int>& frames_, const std::vector<int>& layers_,
                                    const Kernel2D& kernel_, float strength, bool filterAlpha)
    {
        DMXImage out_tmp = in_;

        int framesSize = frames_.size();
        int layersSize = layers_.size();
        int kernelSize = kernel_.size();
        std::size_t N = in_.data().size();

        int* d_frames = nullptr;
        int* d_layers = nullptr;
        float* d_kernel = nullptr;
        float* d_in = nullptr;
        float* d_out = nullptr;
        
        DMXImageView in;
        in.width = in_.width();
        in.height = in_.height();
        in.numLayers = in_.numLayers();
        in.numFrames = in_.numFrames();
        in.numChannels = in_.numChannels();
        DMXImageView out = in;

        CUDA_CHECK(hipMalloc(&d_frames, framesSize*sizeof(int)));
        CUDA_CHECK(hipMalloc(&d_layers, layersSize*sizeof(int)));
        CUDA_CHECK(hipMalloc(&d_kernel, kernelSize*kernelSize*sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_in, N*sizeof(float)));
        CUDA_CHECK(hipMalloc(&d_out, N*sizeof(float)));

        CUDA_CHECK(hipMemcpy(d_frames, frames_.data(), framesSize*sizeof(int), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_layers, layers_.data(), layersSize*sizeof(int), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_kernel, kernel_.m_data.data(), kernelSize*kernelSize*sizeof(float), hipMemcpyHostToDevice));
        CUDA_CHECK(hipMemcpy(d_in, in_.data().data(), N*sizeof(float), hipMemcpyHostToDevice));

        in.data = d_in;
        out.data = d_out;

        dim3 threads(16, 16, 1);
        dim3 blocks( 
            (in_.width() + threads.x - 1)/threads.x,
            (in_.height() + threads.y - 1)/threads.y,
            std::max(1, framesSize * layersSize)
         );
        
        convolve2D_CUDA_kernel<<<blocks, threads>>>(in, out, d_frames, framesSize, d_layers, 
            layersSize, d_kernel, kernelSize, strength, filterAlpha);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
        
        CUDA_CHECK(hipMemcpy(out_tmp.data().data(), out.data, N*sizeof(float), hipMemcpyDeviceToHost));

        out_ = std::move(out_tmp);

        hipFree(d_in);
        hipFree(d_out);
        hipFree(d_kernel);
        hipFree(d_frames);
        hipFree(d_layers);
    }

} // namespace dmxdenoiser
