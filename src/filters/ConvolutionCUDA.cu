#include "hip/hip_runtime.h"
#include <dmxdenoiser/DMXImageView.hpp>
#include <dmxdenoiser/Pixel.hpp>

#include <hip/hip_runtime.h>

__global__ void convolve2D_CUDA(DMXImageView in, DMXImageView out, int* frames, int framesSize, int* layers, int layersSize,
                                float* kernel, int kernelSize) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    int s = blockIdx.z;

    int frameIdx = layers[s / layersSize];
    int layerIdx = layers[s % layersSize];
    int frame = frames[frameIdx];
    int layer = layers[layerIdx];
    int offset = kernelSize/2;

    PixelRGBA orig = in.get(x, y, frame, layer);
    PixelRGBA sum = {0.0f, 0.0f, 0.0f, 0.0f};
    for(int ky = -offset; ky <= offset; ++ky)
        for(int kx = -offset; kx <= offset; ++kx)
        {
            int px = std::clamp(to_int(x) + kx, 0, width - 1);
            int py = std::clamp(to_int(y) + ky, 0, height - 1);
            sum += m_kernel(ky + offset, kx + offset) * input.get(px, py, frame, layer);
        }
    sum = blendPixels(orig, sum, m_strength, m_filterAlpha);
    output.at(to_int(x), to_int(y), frame, layer) = sum;
     
}
