#include "hip/hip_runtime.h"
#include <dmxdenoiser/DMXImageView.hpp>
#include <dmxdenoiser/Pixel.hpp>

#include <hip/hip_runtime.h>

namespace dmxdenoiser
{

    __global__ void convolve2D_CUDA(DMXImageView in, DMXImageView out, int* frames, int framesSize, int* layers, int layersSize,
                                    float* kernel, int kernelSize, float strength, bool filterAlpha) 
    {
        int x = blockIdx.x * blockDim.x + threadIdx.x;
        int y = blockIdx.y * blockDim.y + threadIdx.y;
        int s = blockIdx.z;

        int frameIdx = layers[s / layersSize];
        int layerIdx = layers[s % layersSize];
        int frame = frames[frameIdx];
        int layer = layers[layerIdx];
        int offset = kernelSize/2;

        PixelRGBA orig = in.get(x, y, frame, layer);
        PixelRGBA sum = {0.0f, 0.0f, 0.0f, 0.0f};
        for(int ky = -offset; ky <= offset; ++ky)
            for(int kx = -offset; kx <= offset; ++kx)
            {
                int px = clampf(x + kx, 0, in.width - 1);
                int py = clampf(y + ky, 0, in.height - 1);
                sum += kernel[(ky + offset)*kernelSize + (kx + offset)] * in.get(px, py, frame, layer);
            }
        sum = blendPixels(orig, sum, strength, filterAlpha);
        out.at(x, y, frame, layer) = sum;
    }

} // namespace dmxdenoiser
